#include "hip/hip_runtime.h"
//
//  main.cpp
//  
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <stdio.h>
#include <assert.h>
#include <vector>
using namespace std;

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_error_handling.h"

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float *a, float *b, float *c, int N, int offset)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (offset + idx < N)
    {
        c[offset + idx] = a[offset + idx] + b[offset + idx];
    }
}

void sample_vec_add(int size = 1048576)
{
    int n = size;
    
    int nBytes = n*sizeof(int);
    
    float *a, *b;  // host data
    float *c;  // results
    
    a = (float *)malloc(nBytes);
    b = (float *)malloc(nBytes);
    c = (float *)malloc(nBytes);
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    printf("Allocating device memory on host..\n");
    
    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    
    printf("Copying to device..\n");
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);
    
    printf("Doing GPU Vector add\n");
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n, 0);
    hipMemcpy(c,c_d,n*sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

void streams_vec_add(int size = 1048576, int n_streams = 0)
{
    int n = size;
    int nBytes = n*sizeof(int);
    
    float *a, *b;  // host data
    float *c;  // results
    
    hipHostAlloc( (void**) &a, n * sizeof(float) ,hipHostMallocDefault );
    hipHostAlloc( (void**) &b, n * sizeof(float) ,hipHostMallocDefault );
    hipHostAlloc( (void**) &c, n * sizeof(float) ,hipHostMallocDefault );
    
    float *a_d,*b_d,*c_d;
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    printf("Allocating device memory on host..\n");

    hipMalloc((void **)&a_d,n*sizeof(float));
    printf("here\n");
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));

    printf("Copying to device..\n");
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    printf("Doing GPU Vector add\n");
    
    //vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);
    
    const int NbStreams = n_streams;
    const int StreamSize = n / NbStreams;
    vector<hipStream_t> Stream(NbStreams);
    for ( int i = 0; i < NbStreams; i++ )
        SAFE_CALL(hipStreamCreate(&Stream[i]));
    
    for ( int i = 0; i < NbStreams; i++ )
    {
        int Offset = i * StreamSize;
        
        hipMemcpyAsync(&a_d[Offset], &a[Offset], StreamSize * sizeof(float),
                                  hipMemcpyHostToDevice, Stream[i]);
        hipMemcpyAsync(&b_d[Offset], &b[Offset], StreamSize * sizeof(float),
                                  hipMemcpyHostToDevice, Stream[i]);
//        hipMemcpyAsync(&c_d[Offset], &c[Offset], StreamSize * sizeof(float),
//                                  hipMemcpyHostToDevice, Stream[i]);
        
        dim3 block(1024);
        dim3 grid((StreamSize - 1)/1024 + 1);
        vectorAddGPU<<<grid, block, 0, Stream[i]>>>(a_d, b_d, c_d, StreamSize, Offset);

//        hipMemcpyAsync(&a[Offset], &a_d[Offset], StreamSize * sizeof(float),
//                                  hipMemcpyDeviceToHost, Stream[i]);
//        hipMemcpyAsync(&b[Offset], &b_d[Offset], StreamSize * sizeof(float),
//                                  hipMemcpyDeviceToHost, Stream[i]);
        hipMemcpyAsync(&c[Offset], &c_d[Offset], StreamSize * sizeof(float),
                                  hipMemcpyDeviceToHost, Stream[i]);

    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
}


int main(int argc, char **argv)
{
    if (argc < 3) {
        printf("Incorrect number of arguments\n");
    }
    sample_vec_add(atoi(argv[1]));
    streams_vec_add(atoi(argv[1]), atoi(argv[2]));

    return 0;
}
