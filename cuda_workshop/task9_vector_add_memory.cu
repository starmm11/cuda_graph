#include "hip/hip_runtime.h"
//
//  main.cpp
//  
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_error_handling.h"

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void unified_sample(int size = 1048576)
{
    printf("Unified device memory\n");
    int n = size;

    int nBytes = n*sizeof(float);

    float *a, *b;  // host data
    float *c;  // results

    printf("Allocating unified device memory on host..\n");
    SAFE_CALL(hipMallocManaged((void **)&a,n*sizeof(float)));
    SAFE_CALL(hipMallocManaged((void **)&b,n*sizeof(float)));
    SAFE_CALL(hipMallocManaged((void **)&c,n*sizeof(float)));

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    hipEvent_t start, stop;
    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));

    SAFE_CALL(hipEventRecord(start));

    printf("Doing GPU Vector add\n");

    SAFE_KERNEL_CALL((vectorAddGPU<<<grid, block>>>(a, b, c, n)));

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));
    float milliseconds = 0;
    SAFE_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    printf("time: %f ms\n\n", milliseconds);

    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipFree(a));
    SAFE_CALL(hipFree(b));
    SAFE_CALL(hipFree(c));
}

void unified_sample_prefetch(int size = 1048576)
{
    printf("Unified device memory PREFETCH\n");
    int n = size;

    int nBytes = n*sizeof(float);

    float *a, *b;  // host data
    float *c;  // results


    printf("Allocating unified device memory on host..\n");
    hipEvent_t start_alloc, stop_alloc;
    SAFE_CALL(hipEventCreate(&start_alloc));
    SAFE_CALL(hipEventCreate(&stop_alloc));
    SAFE_CALL(hipEventRecord(start_alloc));

    SAFE_CALL(hipMallocManaged((void **)&a,n*sizeof(float)));
    SAFE_CALL(hipMallocManaged((void **)&b,n*sizeof(float)));
    SAFE_CALL(hipMallocManaged((void **)&c,n*sizeof(float)));
    SAFE_CALL(hipEventRecord(stop_alloc));
    SAFE_CALL(hipEventSynchronize(stop_alloc));
    float ms_alloc = 0;
    SAFE_CALL(hipEventElapsedTime(&ms_alloc, start_alloc, stop_alloc));

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    int device;
    hipGetDevice ( &device );
    printf ( "Device %d\n", device );
    hipEvent_t start, start_kernel, stop;
    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&start_kernel));
    SAFE_CALL(hipEventCreate(&stop));

    SAFE_CALL(hipEventRecord(start));
    printf("Prefetch unified device memory on host..\n");
    SAFE_CALL(hipMemPrefetchAsync ( a, n*sizeof(float), device ));
    SAFE_CALL(hipMemPrefetchAsync ( b, n*sizeof(float), device ));
    SAFE_CALL(hipMemPrefetchAsync ( c, n*sizeof(float), device ));

    printf("Doing GPU Vector add\n");
    SAFE_CALL(hipEventRecord(start_kernel));
    SAFE_KERNEL_CALL((vectorAddGPU<<<grid, block>>>(a, b, c, n)));

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));
    float milliseconds = 0, milliseconds_kernel;
    SAFE_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    SAFE_CALL(hipEventElapsedTime(&milliseconds_kernel, start_kernel, stop));
    printf("allocation time: %f ms\n", ms_alloc);
    printf("time: %f ms\n", milliseconds);
    printf("time kernel: %f ms\n\n", milliseconds_kernel);

    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipFree(a));
    SAFE_CALL(hipFree(b));
    SAFE_CALL(hipFree(c));
}

void pinned_sample(int size = 1048576)
{
    printf("PINNED device memory\n");
    int n = size;

    int nBytes = n*sizeof(float);

    float *a, *b;  // host data
    float *c;  // results
    hipEvent_t start_alloc, stop_alloc;
    SAFE_CALL(hipEventCreate(&start_alloc));
    SAFE_CALL(hipEventCreate(&stop_alloc));
    SAFE_CALL(hipEventRecord(start_alloc));

    SAFE_CALL(hipHostMalloc((void **)&a,n*sizeof(float)));
    SAFE_CALL(hipHostMalloc((void **)&b,n*sizeof(float)));
    SAFE_CALL(hipHostMalloc((void **)&c,n*sizeof(float)));
    SAFE_CALL(hipEventRecord(stop_alloc));
    SAFE_CALL(hipEventSynchronize(stop_alloc));
    float ms_alloc = 0;
    SAFE_CALL(hipEventElapsedTime(&ms_alloc, start_alloc, stop_alloc));

    float *a_d,*b_d,*c_d;

    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));

    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }

    printf("Allocating pinned device memory on host..\n");
    SAFE_CALL(hipMalloc((void **)&a_d,n*sizeof(float)));
    SAFE_CALL(hipMalloc((void **)&b_d,n*sizeof(float)));
    SAFE_CALL(hipMalloc((void **)&c_d,n*sizeof(float)));

    printf("Copying to device..\n");
    hipEvent_t start, start_kernel, stop;
    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));

    SAFE_CALL(hipEventRecord(start));
    SAFE_CALL(hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice));

    printf("Doing GPU Vector add\n");
    SAFE_CALL(hipEventCreate(&start_kernel));
    SAFE_CALL(hipEventRecord(start_kernel));
    SAFE_KERNEL_CALL((vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n)));

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));
    float milliseconds = 0, milliseconds_kernel =0;
    SAFE_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    SAFE_CALL(hipEventElapsedTime(&milliseconds_kernel, start_kernel, stop));
    printf("allocation time: %f ms\n", ms_alloc);
    printf("time: %f ms\n", milliseconds);
    printf("time kernel: %f ms\n\n", milliseconds_kernel);

    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipFree(a_d));
    SAFE_CALL(hipFree(b_d));
    SAFE_CALL(hipFree(c_d));
}

void usual_sample(int size = 1048576)
{
    printf("USUAL device memory\n");
    int n = size;
    
    int nBytes = n*sizeof(float);
    
    float *a, *b;  // host data
    float *c;  // results
    hipEvent_t start_alloc, stop_alloc;
    SAFE_CALL(hipEventCreate(&start_alloc));
    SAFE_CALL(hipEventCreate(&stop_alloc));
    SAFE_CALL(hipEventRecord(start_alloc));

    a = (float *)malloc(nBytes);
    b = (float *)malloc(nBytes);
    c = (float *)malloc(nBytes);
    SAFE_CALL(hipEventRecord(stop_alloc));
    SAFE_CALL(hipEventSynchronize(stop_alloc));
    float ms_alloc = 0;
    SAFE_CALL(hipEventElapsedTime(&ms_alloc, start_alloc, stop_alloc));


    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    
    printf("Allocating device memory on host..\n");
    
    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    
    printf("Copying to device..\n");

    hipEvent_t start, start_kernel, stop;
    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&start_kernel));
    SAFE_CALL(hipEventCreate(&stop));

    SAFE_CALL(hipEventRecord(start));
    
    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);
    
    printf("Doing GPU Vector add\n");
    SAFE_CALL(hipEventRecord(start_kernel));
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0, milliseconds_kernel = 0;
    SAFE_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    SAFE_CALL(hipEventElapsedTime(&milliseconds_kernel, start_kernel, stop));
    printf("allocation time: %f ms\n", ms_alloc);
    printf("time: %f ms\n", milliseconds);
    printf("time kernel: %f ms\n\n", milliseconds_kernel);
    
    hipDeviceSynchronize();
    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}


int main(int argc, char **argv)
{
    usual_sample(atoi(argv[1]));
    pinned_sample(atoi(argv[1]));
    unified_sample(atoi(argv[1]));
    unified_sample_prefetch(atoi(argv[1]));
    return 0;
}
