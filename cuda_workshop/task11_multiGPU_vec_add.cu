#include "hip/hip_runtime.h"
//
//  main.cpp
//  
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_error_handling.h"
#include <hipblas.h>
#include <omp.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void sample_vec_add(int size = 1048576)
{
    int n = size;
    
    int nBytes = n*sizeof(int);
    
    float *a, *b;  // host data
    float *c;  // results
    
    a = (float *)malloc(nBytes);
    b = (float *)malloc(nBytes);
    c = (float *)malloc(nBytes);
    float *c_ans = (float *)malloc(nBytes);
    
    float *a_d,*b_d,*c_d;
    
    dim3 block(256);
    dim3 grid((unsigned int)ceil(n/(float)block.x));
    
    for(int i=0;i<n;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
        c_ans[i] = a[i] + b[i];
    }
    
    printf("Allocating device memory on host..\n");
    
    hipMalloc((void **)&a_d,n*sizeof(float));
    hipMalloc((void **)&b_d,n*sizeof(float));
    hipMalloc((void **)&c_d,n*sizeof(float));
    
    printf("Copying to device..\n");
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    hipMemcpy(a_d,a,n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,n*sizeof(float), hipMemcpyHostToDevice);
    
    printf("Doing GPU Vector add\n");
    
    vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(c, c_d, size * sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %f ms\n", milliseconds);
    
    hipDeviceSynchronize();
    for (int i = 0; i < size; ++i) {
        if (c_ans[i] != c[i]) {
            printf("Test is failed\n");
            printf("%f %f", c_ans[i],c[i]);
            return;
        }

    }
    printf("Tests are good\n");
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a);
    free(b);
    free(c);
}

void streams_vec_add(int size = 1048576)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("gpu count: %d\n", deviceCount);
    float *a, *b;  // host data
    float *c;  // results
    float *cpu_ans;
    a = new float[size];
    b = new float[size];
    c = new float[size];
    cpu_ans = new float[size];
    //hipHostRegister(a,size * sizeof(float),0);
    //hipHostRegister(b,size * sizeof(float),0);
    for(int i=0;i<size;i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
        c[i] = 0;
    }
    for(int i=0;i<size;i++)
    {
        cpu_ans[i] = a[i] + b[i];
    }
    double start = omp_get_wtime();
    #pragma omp parallel num_threads(deviceCount)
    {
        int device = omp_get_thread_num();
        printf("device: %d\n", device);
        hipSetDevice(device); // устанавливаем для каждого потока свой контекст
        int elemsPerDevice = size/deviceCount;
        float *a_d,*b_d,*c_d;
        SAFE_CALL(hipMalloc(&a_d, elemsPerDevice * sizeof(float)));
        SAFE_CALL(hipMalloc(&b_d, elemsPerDevice * sizeof(float)));
        SAFE_CALL(hipMalloc(&c_d, elemsPerDevice * sizeof(float)));
        SAFE_CALL(hipMemcpy(a_d, a + device*elemsPerDevice, elemsPerDevice * sizeof(float), hipMemcpyHostToDevice));
        SAFE_CALL(hipMemcpy(b_d, b + device*elemsPerDevice, elemsPerDevice * sizeof(float), hipMemcpyHostToDevice));
        int blockSize = 128;
        int gridSize = (elemsPerDevice-1)/ blockSize + 1;
        SAFE_KERNEL_CALL((vectorAddGPU<<<gridSize, blockSize>>>(a_d, b_d, c_d, elemsPerDevice)));
        SAFE_CALL(hipMemcpy(c + device*elemsPerDevice, c_d, elemsPerDevice * sizeof(float), hipMemcpyDeviceToHost));
        SAFE_CALL(hipDeviceSynchronize());
        SAFE_CALL(hipDeviceSynchronize());
    }
    double end = omp_get_wtime();
    printf("time %f\n", (end - start)*1e4);
    for (int i = 0; i < size; ++i) {
        if (cpu_ans[i] != c[i]) {
            printf("Test is failed\n");
            printf("%f %f\n", cpu_ans[i],c[i]);
            return;
        }

    }
    printf("Tests are good\n");
}


int main(int argc, char **argv)
{
    sample_vec_add(atoi(argv[1]));
    streams_vec_add(atoi(argv[1]));

    return 0;
}
